#include "hip/hip_runtime.h"
#include <THC/THC.h>

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit
//#define FLT_MAX 3.40282347E+38F

extern "C"
{
void SpatialAdaptiveMaxPooling_updateOutput(THCudaTensor* input, THCudaTensor* output, THCudaTensor* indices, int kW, int kH);
}


/*
 * Description:
 *    this function maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y 
 */
__global__ void adaptivemaxpool(float *input, float *output, float *indices_x, float *indices_y,
                        int input_n, int input_h, int input_w,
			int output_h, int output_w)
                        //int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // output size
  //const int output_w = kW;//(input_w - kW) / dW + 1;
  //const int output_h = kH;//(input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  const int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  const int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);
    int y_end   = (int)ceil(float(yy+1) / output_h * input_h);
    int kH = y_end-y_start;

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      int x_start = (int)floor(float(xx) / output_w * input_w);
      int x_end   = (int)ceil(float(xx + 1) / output_w * input_w);

      int kW = x_end-x_start;

      // Compute the mean of the input image...
      //float *ptr_input = input + yy*dH*input_w + xx*dW;
      float *ptr_input = input + y_start*input_w + x_start;

      float *ptr_output = output + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      int argmax_x = -1;
      int argmax_y = -1;
      float max = -FLT_MAX;
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++) {
          float val = ptr_input[kx];
          if (val > max) {
            max = val;
            argmax_x = kx;
            argmax_y = ky;
          } 
        }
        ptr_input += input_w; // next input line
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind_x = argmax_x + 1;
      *ptr_ind_y = argmax_y + 1;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 */
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y,
                             int input_n, int input_h, int input_w,
                             int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // output size
  int output_w = (input_w - kW) / dW + 1;
  int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      float z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x)-1;
      int argmax_y = (*ptr_ind_y)-1;

      ptr_gradInput[argmax_x + argmax_y*input_w] += z;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 *    when kH != dH or kW != dW (uses atomic add)
 */
__global__ void atomicadaptivemaxgradinput(
  float *gradInput, float *gradOutput, float *indices_x, float *indices_y,
  int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW
)
{
  // iterators
  int xx, yy;

  // output size
  int output_w = (input_w - kW) / dW + 1;
  int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      float z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x)-1;
      int argmax_y = (*ptr_ind_y)-1;

      // atomic add since different threads could update same variable
      atomicAdd(&(ptr_gradInput[argmax_x + argmax_y*input_w]), z);
    }
  }
}

//static int cunn_SpatialAdaptiveMaxPooling_updateOutput(lua_State *L)
//{
//  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
//  int kW = luaT_getfieldcheckint(L, 1, "kW");
//  int kH = luaT_getfieldcheckint(L, 1, "kH");
  //int dW = luaT_getfieldcheckint(L, 1, "dW");
  //int dH = luaT_getfieldcheckint(L, 1, "dH");
void SpatialAdaptiveMaxPooling_updateOutput(THCudaTensor* input, THCudaTensor* output, THCudaTensor* indices, int kW, int kH)
{
//  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
//  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  float *indices_data;
  float *output_data;
  float *input_data;

  //luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];
    long nOutputCols = kW;
    long nOutputRows = kH;

    //luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(input);
    input_data = THCudaTensor_data(input);

    THCudaTensor_resize3d(output, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_resize4d(indices, 2, nInputPlane, nOutputRows, nOutputCols);
    
    indices_data = THCudaTensor_data(indices);
    output_data = THCudaTensor_data(output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run maxpool kernel
    adaptivemaxpool <<<blocks, threads>>> (input_data, output_data, 
                                   indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                   nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];
    long nOutputCols = kW;
    long nOutputRows = kH;

    //luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(input);
    input_data = THCudaTensor_data(input);

    THCudaTensor_resize4d(output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_resize5d(indices, 2, nbatch, nInputPlane, nOutputRows, nOutputCols);

    indices_data = THCudaTensor_data(indices);
    output_data = THCudaTensor_data(output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run maxpool kernel
    adaptivemaxpool <<<blocks, threads>>> (input_data, output_data,
                                   indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                   nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
  }

  // clean
  THCudaTensor_free(input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxsampling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  //return 1;
}
